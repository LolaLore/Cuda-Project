#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include "CImg.h"

using namespace std;
using namespace cimg_library;


__global__ void convertToBlue
	(
	unsigned char *sourceFile,		// picture sourse file
	unsigned char *destinationFile,	// picture destination file
	int width,						// picture width
	int height						// picture height
	)
{
	int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
	int pos_y = blockIdx.y * blockDim.y + threadIdx.y;

	if (pos_x >= width || pos_y >= height)
		return;

	unsigned char r = sourceFile[pos_y * width + pos_x];
	unsigned char g = sourceFile[(height + pos_y) * width + pos_x];
	unsigned char b = sourceFile[(height * 2 + pos_y) * width + pos_x];

	// convert the color
	//unsigned int _gray = (unsigned int)(0.21f * r + 0.71f * g + 0.07f * b);
	//unsigned int _gray = (unsigned int)((100*b/(1+r+g))*256/(1+b+g+r));
	//r = 255 - r;
	//g = 255 - g;
	//b = 255 - b;
	unsigned int _gray = (unsigned int)((r + g + b));
	unsigned char gray = _gray > 255 ? 255 : _gray;
	//printf("%d %d %d\n", r, g, b);

	// commpose the output image
	//destinationFile[pos_y * width + pos_x] = 255-r;
	//destinationFile[(height + pos_y) * width + pos_x]=255-g;
	//destinationFile[(height * 2 + pos_y) * width + pos_x]=255-b;
	destinationFile[pos_y * width + pos_x] = b;
}

int main()
{
	CImg<unsigned char> src("D:\\Facultate_semestrul_V\\PPD\\lab5_ppd_cuda_proiect\\lab5_ppd_cuda_proiect\\pictures\\2.ppm");
	int width = src.width();
	int height = src.height();
	unsigned long size = src.size();

	//create pointer to source image
	unsigned char *srcPointer = src.data();

	CImg<unsigned char> dst(width, height, 1, 3);

	//create pointer to destination image
	unsigned char *dstPointer = dst.data();

	unsigned char *sourceFile;
	unsigned char *destinationFile;

	hipMalloc((void**)&sourceFile, size);
	hipMalloc((void**)&destinationFile, size);

	hipMemcpy(sourceFile, srcPointer, size, hipMemcpyHostToDevice);

	//launch the kernel
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((width + 15) / 16, (height + 15) / 16, 1);
	convertToBlue <<< grdDim, blkDim >>> (sourceFile, destinationFile, width, height);

	//wait until kernel finishes
	hipDeviceSynchronize();

	//copy back the result to CPU
	hipMemcpy(dstPointer, destinationFile, width*height, hipMemcpyDeviceToHost);

	hipFree(sourceFile);
	hipFree(destinationFile);

	CImgDisplay sourceDisplay(src, "Before conversion");
	CImgDisplay destinationDisplay(dst, "After conversion");

	while (!destinationDisplay.is_closed() && !sourceDisplay.is_closed())
	{
		destinationDisplay.wait();
		sourceDisplay.wait();
	}

	return 0;
}